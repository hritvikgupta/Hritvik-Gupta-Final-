
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#define check_Error() {                                                              \
    hipError_t e=hipGetLastError();                                                   \
    if(e!=hipSuccess) {                                                                \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));   \
        hipDeviceReset();                                                              \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}
#define Check_Allocation_Return_Value(a){   \
    if(a==NULL) {                           \
    printf("Allocation Error\n");           \
    hipDeviceReset();                      \
    exit(EXIT_FAILURE);                     \
    }                                       \
}



__global__ void firstkernel(double *A,double *B,double *C,int width, double r){
    int cols = blockIdx.y*blockDim.y+threadIdx.y;
    int rows = blockIdx.x*blockDim.x+threadIdx.x;
    int step;
    double prod_val = 0;
    
    if((cols < (int)(width*r)) &&(rows < (int)(width*r))){
    
      for(step=0;step<width;step++){
          prod_val += A[cols*width+step] * B[step*(int)(width*r)+rows];
      }
      
      C[cols*(int)(width*r)+rows] = prod_val;
    }
}

__global__ void secondKernel(double *A,double *B,double *C,int width, double r){
    int cols = blockIdx.y*blockDim.y+threadIdx.y;
    int rows = blockIdx.x*blockDim.x+threadIdx.x;
    int step;
    double prod_val = 0;
    
    if((cols < (int)(width*r)) && (rows < (int)(width*(1-r)))){
      
      for(step=0;step<width;step++){
          prod_val += A[cols*width+step] * B[step*(int)(width*(1-r))+rows];
      }
      
      C[cols*(int)(width*(1-r))+rows] = prod_val;
    }
}

__global__ void thirdKernel(double *A,double *B,double *C,int width, double r){
    int cols = blockIdx.y*blockDim.y+threadIdx.y;
    int rows = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((cols <(int)(width*(1-r))) && (rows <(int)(width*r))){
      for(step=0;step<width;step++){
          prod_val += A[cols*width+step] * B[step*(int)(width*r)+rows];
      }
      
      
      C[cols*(int)(width*r)+rows] = prod_val;
    }
}

__global__ void fourthKernel(double *A,double *B,double *C,int width, double r){
    int cols = blockIdx.y*blockDim.y+threadIdx.y;
    int rows = blockIdx.x*blockDim.x+threadIdx.x;
    
    int step;
    double prod_val = 0;
    if((cols < (int)(width*(1-r))) && (rows < (int)(width*(1-r)))){
      
      for(step=0;step<width;step++){
          prod_val += A[cols*width+step] * B[step*(int)(width*(1-r))+rows];
      }
      C[cols*(int)(width*(1-r))+rows] = prod_val;
    }
}



int main(int argc,char *argv[]){
    const int numStreams = 4;
    hipStream_t streams[numStreams];
    int N = 2048;
    double *hA,*hB,*hC;
    int id,j,i;
    int Envs;
    double r = 0.5;
    double reverse = (1-r);
    double *hA1,*hA2,*hB1,*hB2,*hC1,*hC2,*hC3,*hC4;
    double *dA1,*dA1_2,*dA2,*dA2_2,*dB1,*dB1_2,*dB2,*dB2_2;
    double *dC1,*dC2,*dC3,*dC4;
         
    hipGetDeviceCount(&Envs);     
    hipHostMalloc(&hA,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA)
    hipHostMalloc(&hB,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB)
    hipHostMalloc(&hC,N*N*sizeof(double), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC)
    memset(hC,0,N*N*sizeof(double)); 
    srand (time(NULL));
    
    for(i=0;i<N*N;i++){
        hA[i] = rand()%10;
        hB[i] = rand()%10;
    }
    

    int widthGrid= 1+N/32;
    dim3 dimGrid(widthGrid,widthGrid,1);
    dim3 dimBlock(32,32,1);
   
    id=0;
    hipSetDevice((int)(id%Envs));
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    
    hipHostMalloc(&hA1,(int)(N*N*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA1)
    hipHostMalloc(&hB1,(int)(N*N*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB1)
    hipHostMalloc(&hC1,(int)(N*N*r*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC1)
    
    for(int i=0;i<(int)(N*r);i++){
        for(int j=0;j<N;j++){
            hA1[i*N+j] =  hA[i*N+j];
        }
    }
    for(int i=0;i<N;i++){
        for(int j=0;j<(N*r);j++){
            hB1[i*(int)(N*r)+j] =  hB[i*N+j];
        }
    }

    hipMalloc((void**)&dA1,(int)(N*N*r*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dB1,(int)(N*N*r*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dC1,(int)(N*N*r*r*sizeof(double)));
    check_Error()
    
    id=1;
    hipSetDevice((int)(id%Envs));
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    hipHostMalloc(&hB2,(int)(N*N*reverse*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hB2)
    hipHostMalloc(&hC2,(int)(N*N*r*reverse*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC2)
    
    for(int i=0;i<N;i++){
        for(int j=0;j<(N*reverse);j++){
            hB2[i*(int)(N*reverse)+j] =  hB[i*N+(int)(N*r)+j];
        }
    }
     
    hipMalloc((void**)&dA1_2,(int)(N*N*r*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dB2,(int)(N*N*reverse*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dC2,(int)(N*N*r*reverse*sizeof(double)));
    check_Error()
        
    id=2;
    hipSetDevice(id%Envs);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    hipHostMalloc(&hA2,(int)(N*N*reverse*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hA2)
    hipHostMalloc(&hC3,(int)(N*N*reverse*r*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC3)
    
    for(int i=0;i<(int)(N*reverse);i++){
        for(int j=0;j<N;j++){
            hA2[i*N+j] =  hA[(i+(int)(N*r))*N+j];
        }
    }
    
    hipMalloc((void**)&dA2,(int)(N*N*reverse*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dB1_2,(int)(N*N*r*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dC3,(int)(N*N*r*reverse*sizeof(double)));
    check_Error()  
        

    id=3;
    hipSetDevice(id%Envs);
    hipStreamCreateWithFlags(&streams[id],hipStreamNonBlocking);
    hipHostMalloc(&hC4,(int)(N*N*reverse*reverse*sizeof(double)), hipHostMallocDefault);
    Check_Allocation_Return_Value(hC4)
    hipMalloc((void**)&dA2_2,(int)(N*N*reverse*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dB2_2,(int)(N*N*reverse*sizeof(double)));
    check_Error()
    hipMalloc((void**)&dC4,(int)(N*N*reverse*reverse*sizeof(double)));
    check_Error()
    
    id=0;
    hipSetDevice(id%Envs);
    hipMemcpyAsync(dA1,hA1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    hipMemcpyAsync(dB1,hB1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    firstkernel <<< dimGrid,dimBlock,0,streams[id]>>>(dA1,dB1,dC1,N,r);
    check_Error()

    
    id=1;
    hipSetDevice(id%Envs);
    hipMemcpyAsync(dA1_2,hA1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    hipMemcpyAsync(dB2,hB2,(int)(N*N*reverse*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    secondKernel <<< dimGrid,dimBlock,0,streams[id]>>>(dA1_2,dB2,dC2,N,r);
    check_Error()
    
    
    id=2;
    hipSetDevice(id%Envs);
    hipMemcpyAsync(dA2,hA2,(int)(N*N*reverse*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    hipMemcpyAsync(dB1_2,hB1,(int)(N*N*r*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    thirdKernel <<< dimGrid,dimBlock,0,streams[id]>>>(dA2,dB1_2,dC3,N,r);
    check_Error()
    
    id=3;
    hipSetDevice(id%Envs);
    hipMemcpyAsync(dA2_2,hA2,(int)(N*N*reverse*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    hipMemcpyAsync(dB2_2,hB2,(int)(N*N*reverse*sizeof(double)),hipMemcpyHostToDevice,streams[id]);
    check_Error()
    fourthKernel <<< dimGrid,dimBlock,0,streams[id]>>>(dA2_2,dB2_2,dC4,N,r);
    check_Error()
    hipMemcpyAsync(hC1,dC1,(int)(N*N*r*r*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    check_Error()
    hipMemcpyAsync(hC2,dC2,(int)(N*N*r*reverse*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    check_Error()
    hipMemcpyAsync(hC3,dC3,(int)(N*N*r*reverse*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    check_Error()
    hipMemcpyAsync(hC4,dC4,(int)(N*N*reverse*reverse*sizeof(double)),hipMemcpyDeviceToHost,streams[id]);
    check_Error()

    for(i=0;i<4;i++)
    {
      id = i;
      hipSetDevice(id%Envs);
      hipStreamSynchronize(streams[id]);
    }

    for(i=0;i<(int)N*r;i++){
        for(j=0;j<(int)N*r;j++){
              hC[i*N+j] = hC1[i*(int)(N*r)+j];
        }
    }
    
    
    
    for(i=0;i<(int)N*r;i++){
        for(j=0;j<(int)(N*reverse);j++){
             hC[i*N+j+(int)(N*r)] = hC2[i*(int)(N*reverse)+j];
        }
    }
    
    for(i=0;i<(int)(N*reverse);i++){
        for(j=0;j<(int)(N*r);j++){
             hC[(i+(int)(N*r))*N+j] = hC3[i*(int)(N*r)+j];
        }
    }

    
    for(i=0;i<(int)(N*reverse);i++){
        for(j=0;j<(int)(N*reverse);j++){
            hC[(i+(int)(N*r))*N+j+(int)(N*r)] = hC4[i*(int)(N*reverse)+j];
        }
    }
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipHostFree(hA1);
    hipHostFree(hA2);
    hipHostFree(hB1);
    hipHostFree(hB2);
    hipHostFree(hC1);
    hipHostFree(hC2);
    hipHostFree(hC3);
    hipHostFree(hC4);
    
    id=0;
    hipSetDevice(id%Envs);
    hipFree(dA1);
    check_Error()
    hipFree(dB1);
    check_Error()
    hipFree(dC1);
    check_Error()
    
    id=1;
    hipSetDevice(id%Envs);
    hipFree(dA1_2);
    check_Error()
    hipFree(dB2);
    check_Error()
    hipFree(dC2);
    check_Error()
    
    id=2;
    hipSetDevice(id%Envs);
    hipFree(dA2);
    check_Error()
    hipFree(dB1_2);
    check_Error()
    hipFree(dC3);
    check_Error()
    
    id=3;
    hipSetDevice(id%Envs);
    hipFree(dA2_2);
    check_Error()
    hipFree(dB2_2);
    check_Error()
    hipFree(dC4);
    check_Error()
    
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
    hipStreamDestroy(streams[2]);
    hipStreamDestroy(streams[3]);
    
    return(0);
}